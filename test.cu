
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
    printf("Hello, world from GPU!\n");
}

int main(){
    printf("Hello, world from CPU!\n");

    hello<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}